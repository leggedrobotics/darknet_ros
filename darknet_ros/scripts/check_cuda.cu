
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv){
    hipDeviceProp_t dP;
    float min_cc = 3.0;
    int deviceCount, rc; 
    rc = hipGetDeviceCount(&deviceCount);
    
    if(rc != hipSuccess) {
        hipError_t error = hipGetLastError();
        printf("CUDA error: %s", hipGetErrorString(error));
        return rc; /* Failure */
    }

    for (int i=0; i < deviceCount; i++)
    {
        rc = hipGetDeviceProperties(&dP, 0);
        if(rc != hipSuccess) {
            hipError_t error = hipGetLastError();
            printf("CUDA error: %s", hipGetErrorString(error));
            return rc; /* Failure */
        }
        if((dP.major+(dP.minor/10)) < min_cc) {
            printf("Min Compute Capability of %2.1f required:  %d.%d found\n Not Building CUDA Code", min_cc, dP.major, dP.minor);
            return 1; /* Failure */
        } else {
            int v = dP.major*10 + dP.minor;
            if (i<deviceCount-1)
                printf("-gencode arch=compute_%d,code=sm_%d;",v,v);
            else
                printf("-gencode arch=compute_%d,code=sm_%d",v,v);
        }
    }
    return 0; /* Success */
}
